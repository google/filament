#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm %s -fcuda-include-gpubinary %s -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

// Make sure that all parts of GPU code init/cleanup are there:
// * constant unnamed string with the kernel name
// CHECK: private unnamed_addr constant{{.*}}kernelfunc{{.*}}\00"
// * constant unnamed string with GPU binary
// CHECK: private unnamed_addr constant{{.*}}\00"
// * constant struct that wraps GPU binary
// CHECK: @__cuda_fatbin_wrapper = internal constant { i32, i32, i8*, i8* } 
// CHECK:       { i32 1180844977, i32 1, {{.*}}, i8* null }
// * variable to save GPU binary handle after initialization
// CHECK: @__cuda_gpubin_handle = internal global i8** null
// * Make sure our constructor/destructor was added to global ctor/dtor list.
// CHECK: @llvm.global_ctors = appending global {{.*}}@__cuda_module_ctor
// CHECK: @llvm.global_dtors = appending global {{.*}}@__cuda_module_dtor

// Test that we build the correct number of calls to hipSetupArgument followed
// by a call to hipLaunchByPtr.

// CHECK: define{{.*}}kernelfunc
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipSetupArgument
// CHECK: call{{.*}}hipLaunchByPtr
__global__ void kernelfunc(int i, int j, int k) {}

// Test that we've built correct kernel launch sequence.
// CHECK: define{{.*}}hostfunc
// CHECK: call{{.*}}hipConfigureCall
// CHECK: call{{.*}}kernelfunc
void hostfunc(void) { kernelfunc<<<1, 1>>>(1, 1, 1); }

// Test that we've built a function to register kernels
// CHECK: define internal void @__cuda_register_kernels
// CHECK: call{{.*}}cudaRegisterFunction(i8** %0, {{.*}}kernelfunc

// Test that we've built contructor..
// CHECK: define internal void @__cuda_module_ctor
//   .. that calls __cudaRegisterFatBinary(&__cuda_fatbin_wrapper)
// CHECK: call{{.*}}cudaRegisterFatBinary{{.*}}__cuda_fatbin_wrapper
//   .. stores return value in __cuda_gpubin_handle
// CHECK-NEXT: store{{.*}}__cuda_gpubin_handle
//   .. and then calls __cuda_register_kernels
// CHECK-NEXT: call void @__cuda_register_kernels

// Test that we've created destructor.
// CHECK: define internal void @__cuda_module_dtor
// CHECK: load{{.*}}__cuda_gpubin_handle
// CHECK-NEXT: call void @__cudaUnregisterFatBinary

