#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm %s -o - -fcuda-is-device -triple nvptx-unknown-unknown | FileCheck %s

// Verifies Clang emits correct address spaces and addrspacecast instructions
// for CUDA code.

#include "Inputs/hip/hip_runtime.h"

// CHECK: @i = addrspace(1) global
__device__ int i;

// CHECK: @j = addrspace(4) global
__constant__ int j;

// CHECK: @k = addrspace(3) global
__shared__ int k;

struct MyStruct {
  int data1;
  int data2;
};

// CHECK: @_ZZ5func0vE1a = internal addrspace(3) global %struct.MyStruct zeroinitializer
// CHECK: @_ZZ5func1vE1a = internal addrspace(3) global float 0.000000e+00
// CHECK: @_ZZ5func2vE1a = internal addrspace(3) global [256 x float] zeroinitializer
// CHECK: @_ZZ5func3vE1a = internal addrspace(3) global float 0.000000e+00
// CHECK: @_ZZ5func4vE1a = internal addrspace(3) global float 0.000000e+00
// CHECK: @b = addrspace(3) global float 0.000000e+00

__device__ void foo() {
  // CHECK: load i32, i32* addrspacecast (i32 addrspace(1)* @i to i32*)
  i++;

  // CHECK: load i32, i32* addrspacecast (i32 addrspace(4)* @j to i32*)
  j++;

  // CHECK: load i32, i32* addrspacecast (i32 addrspace(3)* @k to i32*)
  k++;

  static int li;
  // CHECK: load i32, i32* addrspacecast (i32 addrspace(1)* @_ZZ3foovE2li to i32*)
  li++;

  __constant__ int lj;
  // CHECK: load i32, i32* addrspacecast (i32 addrspace(4)* @_ZZ3foovE2lj to i32*)
  lj++;

  __shared__ int lk;
  // CHECK: load i32, i32* addrspacecast (i32 addrspace(3)* @_ZZ3foovE2lk to i32*)
  lk++;
}

__device__ void func0() {
  __shared__ MyStruct a;
  MyStruct *ap = &a; // composite type
  ap->data1 = 1;
  ap->data2 = 2;
}
// CHECK: define void @_Z5func0v()
// CHECK: store %struct.MyStruct* addrspacecast (%struct.MyStruct addrspace(3)* @_ZZ5func0vE1a to %struct.MyStruct*), %struct.MyStruct** %ap

__device__ void callee(float *ap) {
  *ap = 1.0f;
}

__device__ void func1() {
  __shared__ float a;
  callee(&a); // implicit cast from parameters
}
// CHECK: define void @_Z5func1v()
// CHECK: call void @_Z6calleePf(float* addrspacecast (float addrspace(3)* @_ZZ5func1vE1a to float*))

__device__ void func2() {
  __shared__ float a[256];
  float *ap = &a[128]; // implicit cast from a decayed array
  *ap = 1.0f;
}
// CHECK: define void @_Z5func2v()
// CHECK: store float* getelementptr inbounds ([256 x float], [256 x float]* addrspacecast ([256 x float] addrspace(3)* @_ZZ5func2vE1a to [256 x float]*), i32 0, i32 128), float** %ap

__device__ void func3() {
  __shared__ float a;
  float *ap = reinterpret_cast<float *>(&a); // explicit cast
  *ap = 1.0f;
}
// CHECK: define void @_Z5func3v()
// CHECK: store float* addrspacecast (float addrspace(3)* @_ZZ5func3vE1a to float*), float** %ap

__device__ void func4() {
  __shared__ float a;
  float *ap = (float *)&a; // explicit c-style cast
  *ap = 1.0f;
}
// CHECK: define void @_Z5func4v()
// CHECK: store float* addrspacecast (float addrspace(3)* @_ZZ5func4vE1a to float*), float** %ap

__shared__ float b;

__device__ float *func5() {
  return &b; // implicit cast from a return value
}
// CHECK: define float* @_Z5func5v()
// CHECK: ret float* addrspacecast (float addrspace(3)* @b to float*)

struct StructWithCtor {
  __device__ StructWithCtor(): data(1) {}
  __device__ StructWithCtor(const StructWithCtor &second): data(second.data) {}
  __device__ int getData() { return data; }
  int data;
};

__device__ int construct_shared_struct() {
// CHECK-LABEL: define i32 @_Z23construct_shared_structv()
  __shared__ StructWithCtor s;
// CHECK: call void @_ZN14StructWithCtorC1Ev(%struct.StructWithCtor* addrspacecast (%struct.StructWithCtor addrspace(3)* @_ZZ23construct_shared_structvE1s to %struct.StructWithCtor*))
  __shared__ StructWithCtor t(s);
// CHECK: call void @_ZN14StructWithCtorC1ERKS_(%struct.StructWithCtor* addrspacecast (%struct.StructWithCtor addrspace(3)* @_ZZ23construct_shared_structvE1t to %struct.StructWithCtor*), %struct.StructWithCtor* dereferenceable(4) addrspacecast (%struct.StructWithCtor addrspace(3)* @_ZZ23construct_shared_structvE1s to %struct.StructWithCtor*))
  return t.getData();
// CHECK: call i32 @_ZN14StructWithCtor7getDataEv(%struct.StructWithCtor* addrspacecast (%struct.StructWithCtor addrspace(3)* @_ZZ23construct_shared_structvE1t to %struct.StructWithCtor*))
}
