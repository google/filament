#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm %s -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

__global__ void g1(int x) {}

int main(void) {
  // CHECK: call{{.*}}hipConfigureCall
  // CHECK: icmp
  // CHECK: br
  // CHECK: call{{.*}}g1
  g1<<<1, 1>>>(42);
}
