#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -triple nvptx-unknown-unknown -fcuda-allow-host-calls-from-host-device -fcuda-is-device -Wno-cuda-compat -emit-llvm -o - | FileCheck %s

#include "Inputs/hip/hip_runtime.h"

extern "C"
void host_function() {}

// CHECK-LABEL: define void @hd_function_a
extern "C"
__host__ __device__ void hd_function_a() {
  // CHECK: call void @host_function
  host_function();
}

// CHECK: declare void @host_function

// CHECK-LABEL: define void @hd_function_b
extern "C"
__host__ __device__ void hd_function_b(bool b) { if (b) host_function(); }

// CHECK-LABEL: define void @device_function_b
extern "C"
__device__ void device_function_b() { hd_function_b(false); }

// CHECK-LABEL: define void @global_function
extern "C"
__global__ void global_function() {
  // CHECK: call void @device_function_b
  device_function_b();
}

// CHECK: !{{[0-9]+}} = !{void ()* @global_function, !"kernel", i32 1}
