#include "hip/hip_runtime.h"
// Test the Sema analysis of caller-callee relationships of host device
// functions when compiling CUDA code. There are 4 permutations of this test as
// host and device compilation are separate compilation passes, and clang has
// an option to allow host calls from host device functions. __CUDA_ARCH__ is
// defined when compiling for the device and TEST_WARN_HD when host calls are
// allowed from host device functions. So for example, if __CUDA_ARCH__ is
// defined and TEST_WARN_HD is not then device compilation is happening but
// host device functions are not allowed to call device functions.

// RUN: %clang_cc1 -fsyntax-only -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-allow-host-calls-from-host-device -verify %s -DTEST_WARN_HD
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -fcuda-allow-host-calls-from-host-device -verify %s -DTEST_WARN_HD

#include "Inputs/hip/hip_runtime.h"

__host__ void hd1h(void);
#if defined(__CUDA_ARCH__) && !defined(TEST_WARN_HD)
// expected-note@-2 {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
#endif
__device__ void hd1d(void);
#ifndef __CUDA_ARCH__
// expected-note@-2 {{candidate function not viable: call to __device__ function from __host__ __device__ function}}
#endif
__host__ void hd1hg(void);
__device__ void hd1dg(void);
#ifdef __CUDA_ARCH__
__host__ void hd1hig(void);
#if !defined(TEST_WARN_HD)
// expected-note@-2 {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
#endif
#else
__device__ void hd1dig(void); // expected-note {{candidate function not viable: call to __device__ function from __host__ __device__ function}}
#endif
__host__ __device__ void hd1hd(void);
__global__ void hd1g(void); // expected-note {{'hd1g' declared here}}

__host__ __device__ void hd1(void) {
#if defined(TEST_WARN_HD) && defined(__CUDA_ARCH__)
// expected-warning@-2 {{calling __host__ function hd1h from __host__ __device__ function hd1}}
// expected-warning@-3 {{calling __host__ function hd1hig from __host__ __device__ function hd1}}
#endif
  hd1d();
#ifndef __CUDA_ARCH__
// expected-error@-2 {{no matching function}}
#endif
  hd1h();
#if defined(__CUDA_ARCH__) && !defined(TEST_WARN_HD)
// expected-error@-2 {{no matching function}}
#endif

  // No errors as guarded
#ifdef __CUDA_ARCH__
  hd1d();
#else
  hd1h();
#endif

  // Errors as incorrectly guarded
#ifndef __CUDA_ARCH__
  hd1dig(); // expected-error {{no matching function}}
#else
  hd1hig();
#ifndef TEST_WARN_HD
// expected-error@-2 {{no matching function}}
#endif
#endif

  hd1hd();
  hd1g<<<1, 1>>>(); // expected-error {{reference to __global__ function 'hd1g' in __host__ __device__ function}}
}
