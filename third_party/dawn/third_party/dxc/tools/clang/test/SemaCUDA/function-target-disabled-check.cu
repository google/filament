#include "hip/hip_runtime.h"
// Test that we can disable cross-target call checks in Sema with the
// -fcuda-disable-target-call-checks flag. Without this flag we'd get a bunch
// of errors here, since there are invalid cross-target calls present.

// RUN: %clang_cc1 -fsyntax-only -verify %s -fcuda-disable-target-call-checks
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s -fcuda-disable-target-call-checks

// expected-no-diagnostics

#define __device__ __attribute__((device))
#define __global__ __attribute__((global))
#define __host__ __attribute__((host))

__attribute__((host)) void h1();

__attribute__((device)) void d1() {
  h1();
}

__attribute__((host)) void h2() {
  d1();
}

__attribute__((global)) void g1() {
  h2();
}
